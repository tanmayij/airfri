#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include "../include/commit_gpu.cuh"
#include "../include/merkle.cuh"
#include "../include/hash.cuh"
#include "../include/field.cuh"
#define FIELD_WORDS 5
#define HASH_WORDS 4


__device__ void print_field_kernel(const char *label, const uint64_t *field, int field_words) {
    printf("%s: ", label);
    for (int i = 0; i < field_words; i++) {
        printf("%016lx ", field[i]);
    }
    printf("\n");
}

__host__ void print_field_host(const char *label, const uint64_t *field, int field_words) {
    printf("%s: ", label);
    for (int i = 0; i < field_words; i++) {
        printf("%016lx ", field[i]);
    }
    printf("\n");
}

// Kernel for computing layers with codewords up to layer 11
__global__ void commit_kernel(
    uint64_t *device_codeword, uint64_t *device_codeword_nxt, 
    uint64_t *device_alpha, uint64_t *device_offset,
    uint64_t *device_denominator_inv, uint64_t *device_eval_basis,
    uint64_t *device_temp1, uint64_t *device_temp2,
    uint64_t *device_temp3, uint64_t *device_temp4,
    uint64_t *device_temp5, uint64_t *device_alpha_offset, uint64_t *device_layer_hashes,
    uint64_t *device_concatenated_tree, int N, int basis_len, int layer
) {
    int I = blockIdx.x * blockDim.x + threadIdx.x;
    if (I >= N / 2) return;

    int idx1 = 2 * I * FIELD_WORDS;
    int idx2 = (2 * I + 1) * FIELD_WORDS;
    int idx3 = I * FIELD_WORDS;  // Used for codeword computation
    int tree_idx = I * (1 + FIELD_WORDS + HASH_WORDS);  // For [index, hash, element] in concatenated tree

    field_sub(&device_temp1[idx3], &device_codeword[idx1], &device_codeword[idx2], FIELD_WORDS);
    i_th_ele_in_span(&device_alpha_offset[idx3], device_eval_basis, basis_len, 2 * I);
    field_addEqual(&device_alpha_offset[idx3], device_offset, FIELD_WORDS);

    field_sub(&device_temp2[idx3], device_alpha, &device_alpha_offset[idx3], FIELD_WORDS);
    field_mul(&device_temp3[idx3], &device_temp2[idx3], device_denominator_inv, FIELD_WORDS);
    field_mul(&device_temp4[idx3], &device_temp3[idx3], &device_temp1[idx3], FIELD_WORDS);
    field_add(&device_temp5[idx3], &device_temp4[idx3], &device_codeword[idx1], FIELD_WORDS);

    memcpy(&device_codeword_nxt[idx3], &device_temp5[idx3], FIELD_WORDS * sizeof(uint64_t));

    uint64_t combined[FIELD_WORDS + HASH_WORDS];
    uint8_t digest[HASH_SIZE];

    if (layer == 0) {
        // Layer 0: initial hashes of codewords
        memcpy(combined, &device_codeword[idx1], FIELD_WORDS * sizeof(uint64_t));
        memcpy(combined + FIELD_WORDS, &device_codeword[idx2], FIELD_WORDS * sizeof(uint64_t));
        SHA3(digest, (uint8_t *)combined, 2 * FIELD_WORDS * sizeof(uint64_t), 256);
        memcpy(&device_concatenated_tree[tree_idx + 1], digest, HASH_WORDS * sizeof(uint64_t));
    } else if (layer > 0 && layer <= 11) {
        // Intermediate layers: combine codewords and previous layer hashes
        memcpy(combined, &device_codeword[idx1], FIELD_WORDS * sizeof(uint64_t));
        memcpy(combined + FIELD_WORDS, &device_concatenated_tree[tree_idx - HASH_WORDS], HASH_WORDS * sizeof(uint64_t));
        SHA3(digest, (uint8_t *)combined, (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t), 256);
        memcpy(&device_concatenated_tree[tree_idx], digest, HASH_WORDS * sizeof(uint64_t));
        memcpy(&device_concatenated_tree[tree_idx + HASH_WORDS], &device_codeword[idx3], FIELD_WORDS * sizeof(uint64_t));
    }
    __syncthreads();
}

// Kernel for computing remaining layers with only hashes until the Merkle root
__global__ void merkle_kernel(
    uint64_t *device_layer_hashes, 
    uint64_t *device_merkle_root, 
    uint64_t *device_concatenated_tree, 
    int N
) {
    int I = blockIdx.x * blockDim.x + threadIdx.x;

    while (N > 1) {
        if (I < N / 2) {
            uint64_t combined[2 * HASH_WORDS];
            uint8_t digest[HASH_SIZE];

            int idx1 = (2 * I) * HASH_WORDS;
            int idx2 = (2 * I + 1) * HASH_WORDS;
            int idx3 = I * HASH_WORDS;

            memcpy(combined, &device_layer_hashes[idx1], HASH_WORDS * sizeof(uint64_t));
            memcpy(combined + HASH_WORDS, &device_layer_hashes[idx2], HASH_WORDS * sizeof(uint64_t));
            SHA3(digest, (uint8_t *)combined, 2 * HASH_WORDS * sizeof(uint64_t), 256);
            memcpy(&device_layer_hashes[idx3], digest, HASH_WORDS * sizeof(uint64_t));

            int concat_index = I * (1 + HASH_WORDS);
            device_concatenated_tree[concat_index] = idx3;  // Storing index
            memcpy(&device_concatenated_tree[concat_index + 1], digest, HASH_WORDS * sizeof(uint64_t));
        }

        __syncthreads();
        N /= 2;
    }

    if (I == 0) {
        memcpy(device_merkle_root, device_layer_hashes, HASH_SIZE);
        printf("Final Merkle root: ");
        for (int j = 0; j < HASH_WORDS; ++j) {
            printf("%016lx ", device_merkle_root[j]);
        }
        printf("\n");
    }
}


void commit_launch(
    uint64_t **codeword, uint64_t **codeword_nxt, 
    uint64_t *alpha, uint64_t *offset, 
    uint64_t denominator_inv, uint64_t *eval_basis, 
    int N, uint64_t *root, merkleTree *tree 
) {
    printf("Starting commit_launch\n");
    printf("N = %d, FIELD_WORDS = %d\n", N, FIELD_WORDS);
    int basis_len = (int)log2(N);
    printf("basis len: %d\n", basis_len);

    int layer = 17 - (int)log2(N);
//     if(N == 131072) {
//     for (int i = 0; i < N; ++i) {
//         tree[i].index = i + 1;
//         memcpy(tree[i].element, codeword[i], FIELD_WORDS * sizeof(uint64_t));
//         memset(tree[i].hash, 0, HASH_WORDS * sizeof(uint64_t));
//         tree[i].tree_layer = 0;
//     }
// }

    // if (N == 131072) {
    //     initialize_file("temp1.txt");
    //     initialize_file("temp2.txt");
    //     initialize_file("temp3.txt");
    //     initialize_file("temp4.txt");
    //     initialize_file("temp5.txt");
    //     initialize_file("alpha_offset.txt");
    // }

    uint64_t *flattened_codeword = (uint64_t *)malloc(N * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_codeword == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            flattened_codeword[index] = codeword[i][j];
        }
    }

    printf("First few flattened codeword values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_codeword[i]);
    }
    printf("\n");

    uint64_t *flattened_codeword_nxt = (uint64_t *)malloc((N / 2) * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_codeword_nxt == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword_nxt\n");
        free(flattened_codeword);
        return;
    }

    for (int i = 0; i < N / 2; ++i) {
        codeword_nxt[i] = (uint64_t *)malloc(FIELD_WORDS * sizeof(uint64_t));
        if (codeword_nxt[i] == NULL) {
            fprintf(stderr, "Error: malloc failed for codeword_nxt[%d]\n", i);
            for (int j = 0; j < i; ++j) {
                free(codeword_nxt[j]);
            }
            free(flattened_codeword);
            free(flattened_codeword_nxt);
            return;
        }
    }

    uint64_t flattened_eval_basis[basis_len];
    for (int i = 0; i < basis_len; ++i) {
        flattened_eval_basis[i] = eval_basis[i];
    }

    int field_size = N * FIELD_WORDS * sizeof(uint64_t);

    uint64_t *device_codeword, *device_codeword_nxt, *device_alpha, *device_offset;
    uint64_t *device_denominator_inv, *device_eval_basis;
    uint64_t *device_temp1, *device_temp2, *device_temp3, *device_temp4, *device_temp5, *device_alpha_offset;
    uint64_t *device_layer_hashes, *device_merkle_root, *device_concatenated_tree;
    uint64_t *flattened_alpha_offset = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp1 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp2 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp3 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp4 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp5 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));

    flattened_temp1[N/2 * FIELD_WORDS] = {0}, flattened_temp2[N/2 * FIELD_WORDS] = {0}, flattened_temp3[N/2 * FIELD_WORDS] = {0}, flattened_temp4[N/2 * FIELD_WORDS] = {0}, flattened_temp5[N/2 * FIELD_WORDS] = {0};
    flattened_alpha_offset[N/2 * FIELD_WORDS] = {0};
    hipMalloc((void**)&device_codeword, field_size);
    hipMalloc((void**)&device_codeword_nxt, field_size);
    hipMalloc((void**)&device_alpha, FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_offset, FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_denominator_inv, sizeof(uint64_t));
    hipMalloc((void**)&device_eval_basis, basis_len * sizeof(uint64_t));
    hipMalloc((void **)&device_temp1, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp2, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp3, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp4, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp5, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_alpha_offset, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_layer_hashes, N * HASH_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_merkle_root, HASH_SIZE);
    hipMalloc((void**)&device_concatenated_tree, (N / 2) * (1 + FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));

    hipMemcpy(device_codeword, flattened_codeword, field_size, hipMemcpyHostToDevice);
    hipMemcpy(device_alpha, alpha, FIELD_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_offset, offset, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_denominator_inv, &denominator_inv, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_eval_basis, flattened_eval_basis, basis_len * sizeof(uint64_t), hipMemcpyHostToDevice);

    int threads_per_block = 1;
    int num_blocks = (N / 2 + threads_per_block - 1) / threads_per_block;
    commit_kernel<<<num_blocks * 2, threads_per_block>>>(
        device_codeword, device_codeword_nxt, device_alpha, device_offset,
        device_denominator_inv, device_eval_basis, device_temp1, device_temp2,
        device_temp3, device_temp4, device_temp5, device_alpha_offset, device_layer_hashes, device_concatenated_tree, N, basis_len, layer
    );
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        free(flattened_codeword);
        free(flattened_codeword_nxt);
        free(flattened_eval_basis);
        return;
    }
    // Retrieve data from device_concatenated_tree and populate the `tree` structure
    uint64_t *flattened_tree = (uint64_t *)malloc((N / 2) * (1 + FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
    hipMemcpy(flattened_tree, device_concatenated_tree, (N / 2) * (1 + FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_codeword_nxt, device_codeword_nxt, (N / 2) * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp1, device_temp1, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp2, device_temp2, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp3, device_temp3, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp4, device_temp4, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp5, device_temp5, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_alpha_offset, device_alpha_offset, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Populate the `tree` structure with unflattened data
    for (int i = 0; i < N / 2; ++i) {
        int offset = i * (1 + FIELD_WORDS + HASH_WORDS);
        tree[N / 2 + i].index = flattened_tree[offset];  // First element is the index
        memcpy(tree[N / 2 + i].element, &flattened_tree[offset + 1], FIELD_WORDS * sizeof(uint64_t));  // Next FIELD_WORDS elements are the element values
        memcpy(tree[N / 2 + i].hash, &flattened_tree[offset + 1 + FIELD_WORDS], HASH_WORDS * sizeof(uint64_t));  // Next HASH_WORDS elements are the hash
        tree[N / 2 + i].tree_layer = layer + 1;
    }

    // write_to_file("temp1.txt", flattened_temp1, FIELD_WORDS, N/2);
    // write_to_file("temp2.txt", flattened_temp2, FIELD_WORDS, N/2);
    // write_to_file("temp3.txt", flattened_temp3, FIELD_WORDS, N/2);
    // write_to_file("temp4.txt", flattened_temp4, FIELD_WORDS, N/2);
    // write_to_file("temp5.txt", flattened_temp5, FIELD_WORDS, N/2);
    // write_to_file("alpha_offset.txt", flattened_alpha_offset, FIELD_WORDS, N/2);

    if (N == 32) {
        while (N > 1) {
            printf("reaching here tanjan\n");
            int tpb = 32; 
            int nb = 1;
            merkle_kernel<<<nb, tpb>>>(device_layer_hashes, device_merkle_root, device_concatenated_tree, N);
            N = N / 2;
            hipDeviceSynchronize();
            hipMemcpy(root, device_merkle_root, HASH_SIZE, hipMemcpyDeviceToHost);
            for (int i = 0; i < N / 2; ++i) {
                int offset = i * (1 + FIELD_WORDS + HASH_WORDS);
                tree[N / 2 + i].index = flattened_tree[offset];
                memcpy(tree[N / 2 + i].element, &flattened_tree[offset + 1], FIELD_WORDS * sizeof(uint64_t));
                memcpy(tree[N / 2 + i].hash, &flattened_tree[offset + 1 + FIELD_WORDS], HASH_WORDS * sizeof(uint64_t));
                tree[N / 2 + i].tree_layer = layer + 1;
            }
        }
    }

    for (int i = 0; i < N / 2; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            if (index >= (N / 2) * FIELD_WORDS) {
                printf("Out-of-bounds access at index: %d\n", index);
                break;
            }
            codeword_nxt[i][j] = flattened_codeword_nxt[index];
        }
    }

    printf("First few codeword_nxt values:\n");
    for (int i = 0; i < 10; i++) {
        printf("%016lx ", flattened_codeword_nxt[i]);
    }
    printf("\n");

    hipFree(device_codeword);
    hipFree(device_codeword_nxt);
    hipFree(device_alpha);
    hipFree(device_offset);
    hipFree(device_denominator_inv);
    hipFree(device_eval_basis);
    hipFree(device_merkle_root);
    hipFree(device_layer_hashes);
    hipFree(device_concatenated_tree);
    free(flattened_codeword);
    free(flattened_codeword_nxt);
    free(flattened_tree);

    printf("Memory freed and commit_launch completed.\n");
}