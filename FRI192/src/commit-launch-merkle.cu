#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include "../include/hash.cuh"
#include "../include/hash-host.cuh"
#include "../include/commit-launch-merkle.cuh"
#include "../include/field.cuh"
const size_t field_words = 4;

void SHA3_host(uint8_t *hm, const uint8_t *msg, size_t msg_len, size_t bitSize);

__device__ void print_field_kernel(const char *label, const uint64_t *field, int field_words) {
    printf("%s: ", label);
    for (int i = 0; i < field_words; i++) {
        printf("%016lx ", field[i]);
    }
    printf("\n");
}

__host__ void print_field_host(const char *label, const uint64_t *field, int field_words) {
    printf("%s: ", label);
    for (int i = 0; i < field_words; i++) {
        printf("%016lx ", field[i]);
    }
    printf("\n");
}

__host__ void initialize_file(const char *filename) {
    FILE *file = fopen(filename, "w");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        return;
    }
    fclose(file); 
}

__host__ void write_to_file(const char *filename, const uint64_t *data, int field_words, int total_indices) {
    FILE *file = fopen(filename, "a");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        return;
    }

    for (int index = 0; index < total_indices; index++) {
        fprintf(file, "Index %d: ", index);
        for (int i = 0; i < field_words; i++) {
            fprintf(file, "%llx ", (unsigned long long)data[index + i]);
        }
        fprintf(file, "\n");
    }
    fflush(file);  
    fclose(file);
}

__global__ void commit_kernel(
    uint64_t *device_codeword, uint64_t *device_codeword_nxt,
    uint64_t *device_alpha, uint64_t *device_offset,
    uint64_t *device_denominator_inv, uint64_t *device_eval_basis,
    uint64_t *device_temp1, uint64_t *device_temp2,
    uint64_t *device_temp3, uint64_t *device_temp4,
    uint64_t *device_temp5, uint64_t *device_alpha_offset, 
    uint64_t *device_layer_hashes, uint64_t *device_tree_layer, uint64_t *device_tree_layer_nxt, int N, int basis_len
) {
    size_t I = blockIdx.x * blockDim.x + threadIdx.x;

    if (I >= N / 2) return;

    int idx1 = 2 * I * FIELD_WORDS;
    int idx2 = (2 * I + 1) * FIELD_WORDS;
    int idx3 = I * FIELD_WORDS;

    field_sub(&device_temp1[idx3], &device_codeword[idx1], &device_codeword[idx2], field_words);
    i_th_ele_in_span(&device_alpha_offset[idx3], device_eval_basis, basis_len, 2 * I);
    field_addEqual(&device_alpha_offset[idx3], device_offset, field_words); 

    field_sub(&device_temp2[idx3], device_alpha, &device_alpha_offset[idx3], field_words);
    field_mul(&device_temp3[idx3], &device_temp2[idx3], device_denominator_inv, field_words);
    field_mul(&device_temp4[idx3], &device_temp3[idx3], &device_temp1[idx3], field_words);
    field_add(&device_temp5[idx3], &device_temp4[idx3], &device_codeword[idx1], field_words);

    memcpy(&device_codeword_nxt[idx3], &device_temp5[idx3], field_words * sizeof(uint64_t));
}

__global__ void compute_tree_layers(uint64_t *device_codeword_nxt, uint64_t *device_layer_hashes, uint64_t *device_tree_layer,
uint64_t *device_tree_layer_nxt, uint64_t *device_combined_sibling_codewords, uint64_t *device_concat_codeword_to_hash, uint64_t *device_digest, int N)
{   
    size_t I = blockIdx.x * blockDim.x + threadIdx.x;

    if (I >= N / 2) return;
    if (I < N / 2 && N == 131072) {
        int idx1 = 2 * I * FIELD_WORDS;
        int idx2 = (2 * I + 1) * FIELD_WORDS;
        int idx3 = I * FIELD_WORDS;//for codeword_nxt element
        int idx5 = I * HASH_WORDS; //for hash
        int idx4 = I * (FIELD_WORDS + HASH_WORDS);
        //step 1
        memcpy(&device_combined_sibling_codewords[idx3], &device_tree_layer[idx1], FIELD_WORDS * sizeof(uint64_t));
        memcpy(&device_combined_sibling_codewords[idx3 + FIELD_WORDS], &device_tree_layer[idx2], FIELD_WORDS * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx5], (uint8_t *)&device_combined_sibling_codewords[idx3], 2 * FIELD_WORDS * sizeof(uint64_t), 256);
        //step 3
        memcpy(&device_concat_codeword_to_hash[idx4], &device_codeword_nxt[idx3], FIELD_WORDS * sizeof(uint64_t));
        memcpy(&device_concat_codeword_to_hash[idx4 + FIELD_WORDS], &device_digest[idx5], HASH_WORDS * sizeof(uint64_t));
        //step 4: future-> remove this dependency
        memcpy(&device_tree_layer_nxt[idx4], &device_concat_codeword_to_hash[idx4], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));  
    }

    if(I < N/2 && N < 131072 && N >= 64) {
        int idx1 = 2 * I * (FIELD_WORDS + HASH_WORDS);
        int idx2 = (2 * I + 1) * (FIELD_WORDS + HASH_WORDS);
        int idx3 = I * FIELD_WORDS;//for codeword_nxt element
        int idx5 = I * HASH_WORDS; //for hash
        int idx4 = I * (FIELD_WORDS + HASH_WORDS);
        int idx6 = I * 2 * (FIELD_WORDS + HASH_WORDS);
        //step 1
        memcpy(&device_combined_sibling_codewords[idx6], &device_tree_layer[idx1], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        memcpy(&device_combined_sibling_codewords[idx6 + (FIELD_WORDS + HASH_WORDS)], &device_tree_layer[idx2], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx5], (uint8_t *)&device_combined_sibling_codewords[idx6], 2 * (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t), 256);
        //step3
        memcpy(&device_concat_codeword_to_hash[idx4], &device_codeword_nxt[idx3], FIELD_WORDS * sizeof(uint64_t));
        memcpy(&device_concat_codeword_to_hash[idx4 + FIELD_WORDS], &device_digest[idx5], HASH_WORDS * sizeof(uint64_t));
        //step 4
        memcpy(&device_tree_layer_nxt[idx4], &device_concat_codeword_to_hash[idx4], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));

    }
}
__global__ void merkle_kernel(
    uint64_t *device_layer_hashes, 
    uint64_t *device_merkle_root, 
    uint64_t *device_tree_layer,
    uint64_t *device_tree_layer_nxt,
    uint64_t *device_combined_sibling_codewords,
    uint64_t *device_digest,
    uint64_t *device_combined_sibling_hashes,
    int N
) {
    int I = blockIdx.x * blockDim.x + threadIdx.x;
    int idx1, idx2, idx3, idx4;

    if(I < N/2 && (N == 32 || N == 64)){ //only if N is 32 or N is 64, we use codewords
        idx1 = 2 * I * (FIELD_WORDS + HASH_WORDS);
        idx2 = (2 * I + 1) * (FIELD_WORDS + HASH_WORDS);
        idx3 = I * HASH_WORDS;
        idx4 = I * 2 * (FIELD_WORDS + HASH_WORDS);
        //step 1
        memcpy(&device_combined_sibling_codewords[idx4], &device_tree_layer[idx1], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        memcpy(&device_combined_sibling_codewords[idx4 + (FIELD_WORDS + HASH_WORDS)], &device_tree_layer[idx2], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx3],(uint8_t *)&device_combined_sibling_codewords[idx4], 2 * (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t), 256);
        //step 3 - save all hashes in tree_layer_nxt
        memcpy(&device_tree_layer_nxt[idx3], &device_digest[idx3], HASH_WORDS * sizeof(uint64_t) );
    }

    if(I < N/2 && N < 32) 
    {
        idx1 = 2 * I * (HASH_WORDS);
        idx2 = (2 * I + 1) * (HASH_WORDS);
        idx3 = I * HASH_WORDS;
        idx4 = I * 2 * HASH_WORDS;
        //step 1
        memcpy(&device_combined_sibling_hashes[idx4], &device_tree_layer[idx1], (HASH_WORDS) * sizeof(uint64_t));
        memcpy(&device_combined_sibling_hashes[idx4 + (HASH_WORDS)], &device_tree_layer[idx2], (HASH_WORDS) * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx3], (uint8_t *)&device_combined_sibling_hashes[idx3], 2 * (HASH_WORDS) * sizeof(uint64_t), 256);
        //step 3 - save all hashes in tree_layer_nxt
        memcpy(&device_tree_layer_nxt[idx3], &device_digest[idx3], HASH_WORDS * sizeof(uint64_t) );

    }
    // if (I == 0 && N == 2) {
    //     // Combine the last two sibling hashes into device_combined_sibling_hashes
    //     int idx1 = 0 * HASH_WORDS;            // First sibling hash
    //     int idx2 = 1 * HASH_WORDS;            // Second sibling hash
    //     int idx3 = 0 * HASH_WORDS;            // Output digest index
    //     int idx4 = 0 * 2 * HASH_WORDS;        // Combined hash index
    
    //     // Copy the two sibling hashes into a combined buffer
    //     memcpy(&device_combined_sibling_hashes[idx4], &device_tree_layer[idx1], HASH_WORDS * sizeof(uint64_t));
    //     memcpy(&device_combined_sibling_hashes[idx4 + HASH_WORDS], &device_tree_layer[idx2], HASH_WORDS * sizeof(uint64_t));
    
    //     // Compute the SHA3 hash of the two combined sibling hashes
    //     SHA3((uint8_t *)&device_digest[idx3], (uint8_t *)&device_combined_sibling_hashes[idx4], 2 * HASH_WORDS * sizeof(uint64_t), 256);
    
    //     // Store the computed root in device_merkle_root
    //     memcpy(device_merkle_root, &device_digest[idx3], HASH_WORDS * sizeof(uint64_t));
    // }
//     if (I == 0 && N == 2) {
//         // Compute Merkle root within this kernel instead of launching another one
//         int idx1 = 0 * HASH_WORDS; 
//         int idx2 = 1 * HASH_WORDS;
//         int idx3 = 0 * HASH_WORDS;
//         int idx4 = 0 * 2 * HASH_WORDS;

//         uint64_t combined_sibling_hashes[2 * HASH_WORDS];

//         memcpy(&combined_sibling_hashes[0], &device_tree_layer[idx1], HASH_WORDS * sizeof(uint64_t));
//         memcpy(&combined_sibling_hashes[HASH_WORDS], &device_tree_layer[idx2], HASH_WORDS * sizeof(uint64_t));
//         printf("combined_sibling_hashes before SHA3 in commit kernel: ");
//         for (int j = 0; j < 2 * HASH_WORDS; j++) {
//             printf("%016lx ", combined_sibling_hashes[j]);
//         }
//         printf("\n");
//         SHA3((uint8_t *)&device_merkle_root[idx3], (uint8_t *)combined_sibling_hashes, 2 * HASH_WORDS * sizeof(uint64_t), 256);

//         printf("Computed Merkle Root inside kernel: ");
//         for (int j = 0; j < HASH_WORDS; j++) {
//             printf("%016lx ", device_merkle_root[j]);
//         }
//         printf("\n");
//     }
}

__global__ void compute_merkle_root_kernel(
    uint64_t *device_tree_layer,    // Input: layer with two sibling hashes
    uint64_t *device_merkle_root   // Output: the Merkle root
) {
    if (threadIdx.x == 0) {
        // Indices for the two sibling hashes
        int idx1 = 0 * HASH_WORDS;
        int idx2 = 1 * HASH_WORDS;

        // Combined buffer for the two sibling hashes
        uint64_t combined_sibling_hashes[2 * HASH_WORDS];

        // Copy the sibling hashes into the combined buffer
        memcpy(&combined_sibling_hashes[0], &device_tree_layer[idx1], HASH_WORDS * sizeof(uint64_t));
        memcpy(&combined_sibling_hashes[HASH_WORDS], &device_tree_layer[idx2], HASH_WORDS * sizeof(uint64_t));
        printf("combined_sibling_hashes before SHA3 in commit kernel: ");
        for (int j = 0; j < 2 * HASH_WORDS; j++) {
            printf("%016lx ", combined_sibling_hashes[j]);
        }
        printf("\n");
        // Compute the Merkle root using SHA3
        SHA3(
            (uint8_t *)device_merkle_root,                  // Destination: Merkle root
            (uint8_t *)combined_sibling_hashes,            // Source: Combined sibling hashes
            2 * HASH_WORDS * sizeof(uint64_t),             // Input size: 2 sibling hashes
            256                                            // Output size: 256 bits
        );
        printf("Computed Merkle Root inside kernel: ");
        for (int j = 0; j < HASH_WORDS; j++) {
            printf("%016lx ", device_merkle_root[j]);
        }
        printf("\n");
    }
}

void commit_launch(
    uint64_t **codeword, uint64_t **codeword_nxt, 
    uint64_t *alpha, uint64_t *offset, 
    uint64_t denominator_inv, uint64_t *eval_basis, 
    int N, uint64_t *root, uint64_t **tree_layer, uint64_t **tree_layer_nxt, uint64_t ***tree
) {
    printf("Starting commit_launch\n");
    printf("N = %d, FIELD_WORDS = %d\n", N, FIELD_WORDS);
    int basis_len = (int)log2(N);
    printf("basis len: %d\n", basis_len);

    // if (N == 131072) {
    //     initialize_file("temp1.txt");
    //     initialize_file("temp2.txt");
    //     initialize_file("temp3.txt");
    //     initialize_file("temp4.txt");
    //     initialize_file("temp5.txt");
    //     initialize_file("alpha_offset.txt");
    // }

    uint64_t *flattened_codeword = (uint64_t *)malloc(N * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_codeword == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            flattened_codeword[index] = codeword[i][j];
        }
    }

    printf("First few flattened codeword values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_codeword[i]);
    }
    printf("\n");

    uint64_t *flattened_tree_layer;
    if(N == 131072) {
    flattened_tree_layer = (uint64_t *)malloc(N * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_tree_layer == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            flattened_tree_layer[index] = tree_layer[i][j];
        }
    }
    printf("First few flattened tree_layer values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_tree_layer[i]);
    }
    printf("\n");

    } else {
    flattened_tree_layer = (uint64_t *)malloc(N * CONCAT_WORDS * sizeof(uint64_t));
    if (flattened_tree_layer == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < CONCAT_WORDS; ++j) {
            int index = i * CONCAT_WORDS + j;
            flattened_tree_layer[index] = tree_layer[i][j];
        }
    }
    printf("First few flattened tree_layer values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_tree_layer[i]);
    }
    printf("\n");
}


    uint64_t *flattened_codeword_nxt = (uint64_t *)malloc((N / 2) * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_codeword_nxt == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword_nxt\n");
        free(flattened_codeword);
        return;
    }

    for (int i = 0; i < N / 2; ++i) {
        codeword_nxt[i] = (uint64_t *)malloc(FIELD_WORDS * sizeof(uint64_t));
        if (codeword_nxt[i] == NULL) {
            fprintf(stderr, "Error: malloc failed for codeword_nxt[%d]\n", i);
            for (int j = 0; j < i; ++j) {
                free(codeword_nxt[j]);
            }
            free(flattened_codeword);
            free(flattened_codeword_nxt);
            return;
        }
    }

    uint64_t *flattened_tree_layer_nxt = (uint64_t *)malloc((N / 2) * CONCAT_WORDS * sizeof(uint64_t));
    if (flattened_tree_layer_nxt == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_tree_layer_nxt\n");
        free(flattened_codeword);
        return;
    }

    for (int i = 0; i < N / 2; ++i) {
        tree_layer_nxt[i] = (uint64_t *)malloc(CONCAT_WORDS * sizeof(uint64_t));
        if (tree_layer_nxt[i] == NULL) {
            fprintf(stderr, "Error: malloc failed for tree_layer_nxt[%d]\n", i);
            for (int j = 0; j < i; ++j) {
                free(tree_layer_nxt[j]);
            }
            free(flattened_tree_layer_nxt);
            return;
        }
    }


    uint64_t flattened_eval_basis[basis_len];
    for (int i = 0; i < basis_len; ++i) {
        flattened_eval_basis[i] = eval_basis[i];
    }

    int field_size = N * FIELD_WORDS * sizeof(uint64_t);

    uint64_t *device_codeword, *device_codeword_nxt, *device_alpha, *device_offset;
    uint64_t *device_denominator_inv, *device_eval_basis;
    uint64_t *device_temp1, *device_temp2, *device_temp3, *device_temp4, *device_temp5, *device_alpha_offset;
    uint64_t *device_layer_hashes, *device_merkle_root, *device_tree_layer, *device_tree_layer_nxt, *device_combined_sibling_codewords, *device_combined_sibling_hashes, *device_concat_codeword_to_hash, *device_digest;
    uint64_t *flattened_alpha_offset = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp1 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp2 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp3 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp4 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp5 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));

    flattened_temp1[N/2 * FIELD_WORDS] = {0}, flattened_temp2[N/2 * FIELD_WORDS] = {0}, flattened_temp3[N/2 * FIELD_WORDS] = {0}, flattened_temp4[N/2 * FIELD_WORDS] = {0}, flattened_temp5[N/2 * FIELD_WORDS] = {0};
    flattened_alpha_offset[N/2 * FIELD_WORDS] = {0};
    hipMalloc((void**)&device_codeword, field_size);
    hipMalloc((void**)&device_codeword_nxt, (N/2) * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_alpha, FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_offset, FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_denominator_inv, sizeof(uint64_t));
    hipMalloc((void**)&device_eval_basis, basis_len * sizeof(uint64_t));
    hipMalloc((void **)&device_temp1, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp2, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp3, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp4, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp5, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_alpha_offset, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_layer_hashes, N * HASH_WORDS * sizeof(uint64_t));
    if(N == 131072){
    hipMalloc((void**)&device_tree_layer, field_size);
    hipMalloc((void**)&device_combined_sibling_codewords, (N/2) * 2 * FIELD_WORDS * sizeof(uint64_t));
    } else 
    {
        hipMalloc((void**)&device_tree_layer, N * CONCAT_WORDS * sizeof(uint64_t));
        hipMalloc((void**)&device_combined_sibling_codewords, (N/2) * 2 * ( FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
    }
    hipMalloc((void**)&device_tree_layer_nxt, (N/2) * CONCAT_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_merkle_root, HASH_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_concat_codeword_to_hash, (N/2) * (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
    hipMalloc((void**)&device_combined_sibling_hashes, (N/2) * 2 * HASH_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_digest, (N/2) * HASH_WORDS * sizeof(uint64_t));

    hipMemcpy(device_codeword, flattened_codeword, field_size, hipMemcpyHostToDevice);
    hipMemcpy(device_alpha, alpha, FIELD_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_offset, offset, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_denominator_inv, &denominator_inv, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_eval_basis, flattened_eval_basis, basis_len * sizeof(uint64_t), hipMemcpyHostToDevice);
    if(N == 131072) {
    hipMemcpy(device_tree_layer, flattened_tree_layer,  N * FIELD_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    } else 
    {
        hipMemcpy(device_tree_layer, flattened_tree_layer,  N * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    }
    int threads_per_block = 1;
    int num_blocks = (N / 2 + threads_per_block - 1) / threads_per_block;
    commit_kernel<<<num_blocks * 2, threads_per_block>>>(
        device_codeword, device_codeword_nxt, device_alpha, device_offset,
        device_denominator_inv, device_eval_basis, device_temp1, device_temp2,
        device_temp3, device_temp4, device_temp5, device_alpha_offset, device_layer_hashes, device_tree_layer, device_tree_layer_nxt, N, basis_len
    );
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        free(flattened_codeword);
        free(flattened_codeword_nxt);
        // free(flattened_tree_layer);
        //free(flattened_eval_basis);
        return;
    }
    
    compute_tree_layers<<<num_blocks * 2, threads_per_block>>> (
        device_codeword_nxt, device_layer_hashes, device_tree_layer, device_tree_layer_nxt, device_combined_sibling_codewords, device_concat_codeword_to_hash, device_digest, N
    );
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        free(flattened_codeword);
        free(flattened_codeword_nxt);
        // free(flattened_tree_layer);
        //free(flattened_eval_basis);
        return;
    }

    hipMemcpy(flattened_codeword_nxt, device_codeword_nxt, (N / 2) * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp1, device_temp1, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp2, device_temp2, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp3, device_temp3, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp4, device_temp4, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp5, device_temp5, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_alpha_offset, device_alpha_offset, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < N / 2; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            if (index >= (N / 2) * FIELD_WORDS) {
                printf("Out-of-bounds access at index: %d\n", index);
                break;
            }
            codeword_nxt[i][j] = flattened_codeword_nxt[index];
        }
    }

    printf("First few codeword_nxt values:\n");
    for (int i = 0; i < 10; i++) {
        printf("%016lx ", flattened_codeword_nxt[i]);
    }
    printf("\n");
    hipFree(device_codeword);

    for (int i = 0; i < N / 2; ++i) {
        for (int j = 0; j < CONCAT_WORDS; ++j) {
            int index = i * CONCAT_WORDS + j;
            if (index >= (N / 2) * CONCAT_WORDS) {
                printf("Out-of-bounds access at index: %d\n", index);
                break;
            }
            tree_layer_nxt[i][j] = flattened_tree_layer_nxt[index];
        }
    }

    printf("First few tree_layer_nxt values:\n");
    for (int i = 0; i < 10; i++) {
        printf("%016lx ", flattened_tree_layer_nxt[i]);
    }
    printf("\n");
    hipFree(device_tree_layer);
    hipFree(device_tree_layer_nxt);
    if(N!=64){
    free(flattened_tree_layer_nxt);
    }
    // write_to_file("temp1.txt", flattened_temp1, FIELD_WORDS, N/2);
    // write_to_file("temp2.txt", flattened_temp2, FIELD_WORDS, N/2);
    // write_to_file("temp3.txt", flattened_temp3, FIELD_WORDS, N/2);
    // write_to_file("temp4.txt", flattened_temp4, FIELD_WORDS, N/2);
    // write_to_file("temp5.txt", flattened_temp5, FIELD_WORDS, N/2);
    // write_to_file("alpha_offset.txt", flattened_alpha_offset, FIELD_WORDS, N/2);

    if (N == 64) { 
        int tree_idx = 12;  //start with layer 12 for N == 64
        int next_N = N / 2; //initialize to 32 for the next layer size
        
        //first transfer the tree_layer_nxt elements to tree[12]
        tree[tree_idx] = (uint64_t **)malloc((next_N) * sizeof(uint64_t *));
        for (int i = 0; i < next_N; i++) {
            tree[tree_idx][i] = (uint64_t *)malloc(CONCAT_WORDS * sizeof(uint64_t));
            for (int j = 0; j < CONCAT_WORDS; j++) {
                tree[tree_idx][i][j] = flattened_tree_layer_nxt[i * CONCAT_WORDS + j];
            }
        }
        //free this so it can be re-initialized below
        free(flattened_tree_layer_nxt);
        //allocate flattened memory for device -> host transfer
        uint64_t *flattened_tree_layer_nxt = (uint64_t *)malloc((N / 2) * CONCAT_WORDS * sizeof(uint64_t));
    
        //step 1: unflatten tree_layer_nxt computed by commit_kernel and assign it to tree[12]
        hipMalloc((void **)&device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t));
        //hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    
        // tree[tree_idx] = (uint64_t **)malloc((next_N) * sizeof(uint64_t *));
        // for (int i = 0; i < next_N; i++) {
        //     tree[tree_idx][i] = (uint64_t *)malloc(CONCAT_WORDS * sizeof(uint64_t));
        //     for (int j = 0; j < CONCAT_WORDS; j++) {
        //         tree[tree_idx][i][j] = flattened_tree_layer_nxt[i * CONCAT_WORDS + j];
        //     }
        // }
        tree_idx++;  // Move to the next tree layer index

        // Step 2: Assign tree_layer_nxt to tree_layer for the upcoming Merkle kernel computation
        //hipMemcpy(device_tree_layer, device_tree_layer_nxt, (next_N / 2) * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyDeviceToDevice);
        merkle_kernel<<<2, 32>>>(
            device_layer_hashes, 
            device_merkle_root, 
            device_tree_layer, 
            device_tree_layer_nxt, 
            device_combined_sibling_codewords, 
            device_digest, 
            device_combined_sibling_hashes, 
            next_N
        );
        hipDeviceSynchronize();
        //step 3: Loop over remaining layers, updating tree[layer] with each iteration
        while (next_N > 2) {
            int tpb = min(32, next_N / 2);
            if(tpb == 0) {tpb = 1;}
            int nb = (next_N + tpb - 1) / tpb;
            
            // merkle_kernel for each layer (computes next layer hashes)
            merkle_kernel<<<nb, tpb>>>(device_layer_hashes, device_merkle_root, device_tree_layer, device_tree_layer_nxt, device_combined_sibling_codewords, device_digest, device_combined_sibling_hashes, next_N);
            hipDeviceSynchronize();
            
            hipMalloc((void **)&device_tree_layer_nxt, (next_N / 2) * HASH_WORDS * sizeof(uint64_t));
            hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (next_N / 2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
            printf("First few tree_layer_nxt values in the looooop:\n");
            for (int i = 0; i < HASH_WORDS + 4; i++) {
                printf("%016lx ", flattened_tree_layer_nxt[i]);
            }
            printf("\n");
            //unflatten and store in tree[tree_idx]
            tree[tree_idx] = (uint64_t **)malloc((next_N / 2) * sizeof(uint64_t *));
            for (int i = 0; i < next_N / 2; i++) {
                tree[tree_idx][i] = (uint64_t *)malloc(HASH_WORDS * sizeof(uint64_t));
                for (int j = 0; j < HASH_WORDS; j++) {
                    tree[tree_idx][i][j] = flattened_tree_layer_nxt[i * HASH_WORDS + j];
                }
            }
            printf("Populated tree[%d] with %d elements\n", tree_idx, next_N/2);
            tree_idx++;
            
            //step 4: Update device_tree_layer with the contents of device_tree_layer_nxt
            hipMemcpy(device_tree_layer, device_tree_layer_nxt, (next_N / 2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToDevice);

            next_N /= 2;
        // Handle the final layer (tree[16]) to compute the Merkle root
        // if (next_N == 2) {
        //     // Copy Merkle root directly from device after merkle_kernel execution
        //     hipMemcpy(root, device_merkle_root, HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
        
        //     // Print the computed Merkle root
        //     printf("Computed Merkle Root: ");
        //     for (int i = 0; i < HASH_WORDS; i++) {
        //         printf("%016lx ", root[i]);
        //     }
        //     printf("\n");
        // }
        }// close while
        //do not delete the below code - only comment it
        // for (int layer = 13; layer <= 16; layer++) {
        //     int elements = 1 << (17 - layer);  // Number of elements in this layer
        //     printf("\n=== DEBUG: tree[%d] Elements ===\n", layer);
            
        //     for (int i = 0; i < elements; i++) {
        //         printf("Index %d: ", i);
        //         for (int j = 0; j < HASH_WORDS; j++) {
        //             printf("%016lx ", tree[layer][i][j]);  // Print each hash
        //         }
        //         printf("\n");
        //     }
        // }
        printf("\n=== DEBUG: tree[16] Elements (Last Layer Before Merkle Root) ===\n");
        for (int i = 0; i < 2; i++) {  // Only two elements in the final layer
            printf("Index %d: ", i);
            for (int j = 0; j < HASH_WORDS; j++) {
                printf("%016lx ", tree[16][i][j]);  // Print each hash
            }
            printf("\n");
        }
        compute_merkle_root_kernel<<<1, 1>>>(
            device_tree_layer_nxt,  // The last computed layer (contains 2 hashes)
            device_merkle_root      // Output: The final Merkle root
        );
        hipDeviceSynchronize();
        
        hipMemcpy(root, device_merkle_root, HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
        
        printf("Computed Merkle Root: ");
        for (int i = 0; i < HASH_WORDS; i++) {
            printf("%016lx ", root[i]);
        }
        printf("\n");
    
    hipFree(device_codeword_nxt);
    hipFree(device_alpha);
    hipFree(device_offset);
    hipFree(device_denominator_inv);
    hipFree(device_eval_basis);
    hipFree(device_merkle_root);
    hipFree(device_layer_hashes);
    }
    free(flattened_codeword);
    free(flattened_codeword_nxt);

    printf("Memory freed and commit_launch completed.\n");
}
