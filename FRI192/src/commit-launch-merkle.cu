#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include "../include/hash.cuh"
#include "../include/hash-host.cuh"
#include "../include/commit-launch-merkle.cuh"
#include "../include/field.cuh"
const size_t field_words = 4;
__device__ void print_field_kernel(const char *label, const uint64_t *field, int field_words) {
    printf("%s: ", label);
    for (int i = 0; i < field_words; i++) {
        printf("%016lx ", field[i]);
    }
    printf("\n");
}

__host__ void print_field_host(const char *label, const uint64_t *field, int field_words) {
    printf("%s: ", label);
    for (int i = 0; i < field_words; i++) {
        printf("%016lx ", field[i]);
    }
    printf("\n");
}

__host__ void initialize_file(const char *filename) {
    FILE *file = fopen(filename, "w");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        return;
    }
    fclose(file); 
}

__host__ void write_to_file(const char *filename, const uint64_t *data, int field_words, int total_indices) {
    FILE *file = fopen(filename, "a");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        return;
    }

    for (int index = 0; index < total_indices; index++) {
        fprintf(file, "Index %d: ", index);
        for (int i = 0; i < field_words; i++) {
            fprintf(file, "%llx ", (unsigned long long)data[index + i]);
        }
        fprintf(file, "\n");
    }
    fflush(file);  
    fclose(file);
}

__global__ void commit_kernel(
    uint64_t *device_codeword, uint64_t *device_codeword_nxt,
    uint64_t *device_alpha, uint64_t *device_offset,
    uint64_t *device_denominator_inv, uint64_t *device_eval_basis,
    uint64_t *device_temp1, uint64_t *device_temp2,
    uint64_t *device_temp3, uint64_t *device_temp4,
    uint64_t *device_temp5, uint64_t *device_alpha_offset, 
    uint64_t *device_layer_hashes, uint64_t *device_tree_layer, uint64_t *device_tree_layer_nxt, int N, int basis_len
) {
    size_t I = blockIdx.x * blockDim.x + threadIdx.x;

    if (I >= N / 2) return;

    int idx1 = 2 * I * FIELD_WORDS;
    int idx2 = (2 * I + 1) * FIELD_WORDS;
    int idx3 = I * FIELD_WORDS;

    field_sub(&device_temp1[idx3], &device_codeword[idx1], &device_codeword[idx2], field_words);
    i_th_ele_in_span(&device_alpha_offset[idx3], device_eval_basis, basis_len, 2 * I);
    field_addEqual(&device_alpha_offset[idx3], device_offset, field_words); 

    field_sub(&device_temp2[idx3], device_alpha, &device_alpha_offset[idx3], field_words);
    field_mul(&device_temp3[idx3], &device_temp2[idx3], device_denominator_inv, field_words);
    field_mul(&device_temp4[idx3], &device_temp3[idx3], &device_temp1[idx3], field_words);
    field_add(&device_temp5[idx3], &device_temp4[idx3], &device_codeword[idx1], field_words);

    memcpy(&device_codeword_nxt[idx3], &device_temp5[idx3], field_words * sizeof(uint64_t));
}

__global__ void compute_tree_layers(uint64_t *device_codeword_nxt, uint64_t *device_layer_hashes, uint64_t *device_tree_layer,
uint64_t *device_tree_layer_nxt, uint64_t *device_combined_sibling_codewords, uint64_t *device_concat_codeword_to_hash, uint64_t *device_digest, int N)
{   
    size_t I = blockIdx.x * blockDim.x + threadIdx.x;

    if (I >= N / 2) return;
    if (I < N / 2 && N == 131072) {
        int idx1 = 2 * I * FIELD_WORDS;
        int idx2 = (2 * I + 1) * FIELD_WORDS;
        int idx3 = I * FIELD_WORDS;//for codeword_nxt element
        int idx5 = I * HASH_WORDS; //for hash
        int idx4 = I * (FIELD_WORDS + HASH_WORDS);
        //step 1
        memcpy(&device_combined_sibling_codewords[idx3], &device_tree_layer[idx1], FIELD_WORDS * sizeof(uint64_t));
        memcpy(&device_combined_sibling_codewords[idx3 + FIELD_WORDS], &device_tree_layer[idx2], FIELD_WORDS * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx5], (uint8_t *)&device_combined_sibling_codewords[idx3], 2 * FIELD_WORDS * sizeof(uint64_t), 256);
        //step 3
        memcpy(&device_concat_codeword_to_hash[idx4], &device_codeword_nxt[idx3], FIELD_WORDS * sizeof(uint64_t));
        memcpy(&device_concat_codeword_to_hash[idx4 + FIELD_WORDS], &device_digest[idx5], HASH_WORDS * sizeof(uint64_t));
        //step 4: future-> remove this dependency
        memcpy(&device_tree_layer_nxt[idx4], &device_concat_codeword_to_hash[idx4], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
    
    }

    if(I < N/2 && N < 131072 && N >= 64) {
        int idx1 = 2 * I * (FIELD_WORDS + HASH_WORDS);
        int idx2 = (2 * I + 1) * (FIELD_WORDS + HASH_WORDS);
        int idx3 = I * FIELD_WORDS;//for codeword_nxt element
        int idx5 = I * HASH_WORDS; //for hash
        int idx4 = I * (FIELD_WORDS + HASH_WORDS);
        int idx6 = I * 2 * (FIELD_WORDS + HASH_WORDS);
        //step 1
        memcpy(&device_combined_sibling_codewords[idx6], &device_tree_layer[idx1], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        memcpy(&device_combined_sibling_codewords[idx6 + (FIELD_WORDS + HASH_WORDS)], &device_tree_layer[idx2], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx5], (uint8_t *)&device_combined_sibling_codewords[idx6], 2 * (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t), 256);
        //step3
        memcpy(&device_concat_codeword_to_hash[idx4], &device_codeword_nxt[idx3], FIELD_WORDS * sizeof(uint64_t));
        memcpy(&device_concat_codeword_to_hash[idx4 + FIELD_WORDS], &device_digest[idx5], HASH_WORDS * sizeof(uint64_t));
        //step 4
        memcpy(&device_tree_layer_nxt[idx4], &device_concat_codeword_to_hash[idx4], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));

    }
}
__global__ void merkle_kernel(
    uint64_t *device_layer_hashes, 
    uint64_t *device_merkle_root, 
    uint64_t *device_tree_layer,
    uint64_t *device_tree_layer_nxt,
    uint64_t *device_combined_sibling_codewords,
    uint64_t *device_digest,
    uint64_t *device_combined_sibling_hashes,
    int N
) {
    int I = blockIdx.x * blockDim.x + threadIdx.x;
    int idx1, idx2, idx3, idx4;

    if(I < N/2 && N == 32){ //only if N is 32, we use codewords
        idx1 = 2 * I * (FIELD_WORDS + HASH_WORDS);
        idx2 = (2 * I + 1) * (FIELD_WORDS + HASH_WORDS);
        idx3 = I * HASH_WORDS;
        idx4 = I * 2 * (FIELD_WORDS + HASH_WORDS);
        //step 1
        memcpy(&device_combined_sibling_codewords[idx4], &device_tree_layer[idx1], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        memcpy(&device_combined_sibling_codewords[idx4 + (FIELD_WORDS + HASH_WORDS)], &device_tree_layer[idx2], (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx3],(uint8_t *)&device_combined_sibling_codewords[idx4], 2 * (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t), 256);
        //step 3 - save all hashes in tree_layer_nxt
        memcpy(&device_tree_layer_nxt[idx3], &device_digest[idx3], HASH_WORDS * sizeof(uint64_t) );
    }

    if(I < N/2 && N < 32) 
    {
        idx1 = 2 * I * (HASH_WORDS);
        idx2 = (2 * I + 1) * (HASH_WORDS);
        idx3 = I * HASH_WORDS;
        idx4 = I * 2 * HASH_WORDS;
        //step 1
        memcpy(&device_combined_sibling_hashes[idx4], &device_tree_layer[idx1], (HASH_WORDS) * sizeof(uint64_t));
        memcpy(&device_combined_sibling_hashes[idx4 + (HASH_WORDS)], &device_tree_layer[idx2], (HASH_WORDS) * sizeof(uint64_t));
        //step 2
        SHA3((uint8_t *)&device_digest[idx3], (uint8_t *)&device_combined_sibling_hashes[idx3], 2 * (HASH_WORDS) * sizeof(uint64_t), 256);
        //step 3 - save all hashes in tree_layer_nxt
        memcpy(&device_tree_layer_nxt[idx3], &device_digest[idx3], HASH_WORDS * sizeof(uint64_t) );

    }
    // if (I == 0 && N == 2) {
    //     // Combine the last two sibling hashes into device_combined_sibling_hashes
    //     int idx1 = 0 * HASH_WORDS;            // First sibling hash
    //     int idx2 = 1 * HASH_WORDS;            // Second sibling hash
    //     int idx3 = 0 * HASH_WORDS;            // Output digest index
    //     int idx4 = 0 * 2 * HASH_WORDS;        // Combined hash index
    
    //     // Copy the two sibling hashes into a combined buffer
    //     memcpy(&device_combined_sibling_hashes[idx4], &device_tree_layer[idx1], HASH_WORDS * sizeof(uint64_t));
    //     memcpy(&device_combined_sibling_hashes[idx4 + HASH_WORDS], &device_tree_layer[idx2], HASH_WORDS * sizeof(uint64_t));
    
    //     // Compute the SHA3 hash of the two combined sibling hashes
    //     SHA3((uint8_t *)&device_digest[idx3], (uint8_t *)&device_combined_sibling_hashes[idx4], 2 * HASH_WORDS * sizeof(uint64_t), 256);
    
    //     // Store the computed root in device_merkle_root
    //     memcpy(device_merkle_root, &device_digest[idx3], HASH_WORDS * sizeof(uint64_t));
    // }
    if (I == 0 && N == 2) {
        // Compute Merkle root within this kernel instead of launching another one
        int idx1 = 0 * HASH_WORDS; 
        int idx2 = 1 * HASH_WORDS;
        int idx3 = 0 * HASH_WORDS;
        int idx4 = 0 * 2 * HASH_WORDS;

        uint64_t combined_sibling_hashes[2 * HASH_WORDS];

        memcpy(&combined_sibling_hashes[0], &device_tree_layer[idx1], HASH_WORDS * sizeof(uint64_t));
        memcpy(&combined_sibling_hashes[HASH_WORDS], &device_tree_layer[idx2], HASH_WORDS * sizeof(uint64_t));

        SHA3((uint8_t *)&device_merkle_root[idx3], (uint8_t *)combined_sibling_hashes, 2 * HASH_WORDS * sizeof(uint64_t), 256);

        printf("Computed Merkle Root inside kernel: ");
        for (int j = 0; j < HASH_WORDS; j++) {
            printf("%016lx ", device_merkle_root[j]);
        }
        printf("\n");
    }
}

__global__ void compute_merkle_root_kernel(
    uint64_t *device_tree_layer,    // Input: layer with two sibling hashes
    uint64_t *device_merkle_root   // Output: the Merkle root
) {
    if (threadIdx.x == 0) {
        // Indices for the two sibling hashes
        int idx1 = 0 * HASH_WORDS;
        int idx2 = 1 * HASH_WORDS;

        // Combined buffer for the two sibling hashes
        uint64_t combined_sibling_hashes[2 * HASH_WORDS];

        // Copy the sibling hashes into the combined buffer
        memcpy(&combined_sibling_hashes[0], &device_tree_layer[idx1], HASH_WORDS * sizeof(uint64_t));
        memcpy(&combined_sibling_hashes[HASH_WORDS], &device_tree_layer[idx2], HASH_WORDS * sizeof(uint64_t));

        // Compute the Merkle root using SHA3
        SHA3(
            (uint8_t *)device_merkle_root,                  // Destination: Merkle root
            (uint8_t *)combined_sibling_hashes,            // Source: Combined sibling hashes
            2 * HASH_WORDS * sizeof(uint64_t),             // Input size: 2 sibling hashes
            256                                            // Output size: 256 bits
        );
    }
}

void commit_launch(
    uint64_t **codeword, uint64_t **codeword_nxt, 
    uint64_t *alpha, uint64_t *offset, 
    uint64_t denominator_inv, uint64_t *eval_basis, 
    int N, uint64_t *root, uint64_t **tree_layer, uint64_t **tree_layer_nxt, uint64_t ***tree
) {
    printf("Starting commit_launch\n");
    printf("N = %d, FIELD_WORDS = %d\n", N, FIELD_WORDS);
    int basis_len = (int)log2(N);
    printf("basis len: %d\n", basis_len);

    // if (N == 131072) {
    //     initialize_file("temp1.txt");
    //     initialize_file("temp2.txt");
    //     initialize_file("temp3.txt");
    //     initialize_file("temp4.txt");
    //     initialize_file("temp5.txt");
    //     initialize_file("alpha_offset.txt");
    // }

    uint64_t *flattened_codeword = (uint64_t *)malloc(N * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_codeword == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            flattened_codeword[index] = codeword[i][j];
        }
    }

    printf("First few flattened codeword values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_codeword[i]);
    }
    printf("\n");

    uint64_t *flattened_tree_layer;
    if(N == 131072) {
    flattened_tree_layer = (uint64_t *)malloc(N * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_tree_layer == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            flattened_tree_layer[index] = tree_layer[i][j];
        }
    }
    printf("First few flattened tree_layer values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_tree_layer[i]);
    }
    printf("\n");

    } else {
    flattened_tree_layer = (uint64_t *)malloc(N * CONCAT_WORDS * sizeof(uint64_t));
    if (flattened_tree_layer == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword\n");
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < CONCAT_WORDS; ++j) {
            int index = i * CONCAT_WORDS + j;
            flattened_tree_layer[index] = tree_layer[i][j];
        }
    }
    printf("First few flattened tree_layer values:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%016lx ", flattened_tree_layer[i]);
    }
    printf("\n");
}


    uint64_t *flattened_codeword_nxt = (uint64_t *)malloc((N / 2) * FIELD_WORDS * sizeof(uint64_t));
    if (flattened_codeword_nxt == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_codeword_nxt\n");
        free(flattened_codeword);
        return;
    }

    for (int i = 0; i < N / 2; ++i) {
        codeword_nxt[i] = (uint64_t *)malloc(FIELD_WORDS * sizeof(uint64_t));
        if (codeword_nxt[i] == NULL) {
            fprintf(stderr, "Error: malloc failed for codeword_nxt[%d]\n", i);
            for (int j = 0; j < i; ++j) {
                free(codeword_nxt[j]);
            }
            free(flattened_codeword);
            free(flattened_codeword_nxt);
            return;
        }
    }

    uint64_t *flattened_tree_layer_nxt = (uint64_t *)malloc((N / 2) * CONCAT_WORDS * sizeof(uint64_t));
    if (flattened_tree_layer_nxt == NULL) {
        fprintf(stderr, "Error: malloc failed for flattened_tree_layer_nxt\n");
        free(flattened_codeword);
        return;
    }

    for (int i = 0; i < N / 2; ++i) {
        tree_layer_nxt[i] = (uint64_t *)malloc(CONCAT_WORDS * sizeof(uint64_t));
        if (tree_layer_nxt[i] == NULL) {
            fprintf(stderr, "Error: malloc failed for tree_layer_nxt[%d]\n", i);
            for (int j = 0; j < i; ++j) {
                free(tree_layer_nxt[j]);
            }
            free(flattened_tree_layer_nxt);
            return;
        }
    }


    uint64_t flattened_eval_basis[basis_len];
    for (int i = 0; i < basis_len; ++i) {
        flattened_eval_basis[i] = eval_basis[i];
    }

    int field_size = N * FIELD_WORDS * sizeof(uint64_t);

    uint64_t *device_codeword, *device_codeword_nxt, *device_alpha, *device_offset;
    uint64_t *device_denominator_inv, *device_eval_basis;
    uint64_t *device_temp1, *device_temp2, *device_temp3, *device_temp4, *device_temp5, *device_alpha_offset;
    uint64_t *device_layer_hashes, *device_merkle_root, *device_tree_layer, *device_tree_layer_nxt, *device_combined_sibling_codewords, *device_combined_sibling_hashes, *device_concat_codeword_to_hash, *device_digest;
    uint64_t *flattened_alpha_offset = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp1 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp2 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp3 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp4 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));
    uint64_t *flattened_temp5 = (uint64_t *)malloc(N/2 * FIELD_WORDS * sizeof(uint64_t));

    flattened_temp1[N/2 * FIELD_WORDS] = {0}, flattened_temp2[N/2 * FIELD_WORDS] = {0}, flattened_temp3[N/2 * FIELD_WORDS] = {0}, flattened_temp4[N/2 * FIELD_WORDS] = {0}, flattened_temp5[N/2 * FIELD_WORDS] = {0};
    flattened_alpha_offset[N/2 * FIELD_WORDS] = {0};
    hipMalloc((void**)&device_codeword, field_size);
    hipMalloc((void**)&device_codeword_nxt, (N/2) * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_alpha, FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_offset, FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_denominator_inv, sizeof(uint64_t));
    hipMalloc((void**)&device_eval_basis, basis_len * sizeof(uint64_t));
    hipMalloc((void **)&device_temp1, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp2, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp3, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp4, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_temp5, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void **)&device_alpha_offset, N/2 * FIELD_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_layer_hashes, N * HASH_WORDS * sizeof(uint64_t));
    if(N == 131072){
    hipMalloc((void**)&device_tree_layer, field_size);
    hipMalloc((void**)&device_combined_sibling_codewords, (N/2) * 2 * FIELD_WORDS * sizeof(uint64_t));
    } else 
    {
        hipMalloc((void**)&device_tree_layer, N * CONCAT_WORDS * sizeof(uint64_t));
        hipMalloc((void**)&device_combined_sibling_codewords, (N/2) * 2 * ( FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
    }
    hipMalloc((void**)&device_tree_layer_nxt, (N/2) * CONCAT_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_merkle_root, HASH_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_concat_codeword_to_hash, (N/2) * (FIELD_WORDS + HASH_WORDS) * sizeof(uint64_t));
    hipMalloc((void**)&device_combined_sibling_hashes, (N/2) * 2 * HASH_WORDS * sizeof(uint64_t));
    hipMalloc((void**)&device_digest, (N/2) * HASH_WORDS * sizeof(uint64_t));

    hipMemcpy(device_codeword, flattened_codeword, field_size, hipMemcpyHostToDevice);
    hipMemcpy(device_alpha, alpha, FIELD_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_offset, offset, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_denominator_inv, &denominator_inv, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(device_eval_basis, flattened_eval_basis, basis_len * sizeof(uint64_t), hipMemcpyHostToDevice);
    if(N == 131072) {
    hipMemcpy(device_tree_layer, flattened_tree_layer,  N * FIELD_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    } else 
    {
        hipMemcpy(device_tree_layer, flattened_tree_layer,  N * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyHostToDevice);
    }
    int threads_per_block = 1;
    int num_blocks = (N / 2 + threads_per_block - 1) / threads_per_block;
    commit_kernel<<<num_blocks * 2, threads_per_block>>>(
        device_codeword, device_codeword_nxt, device_alpha, device_offset,
        device_denominator_inv, device_eval_basis, device_temp1, device_temp2,
        device_temp3, device_temp4, device_temp5, device_alpha_offset, device_layer_hashes, device_tree_layer, device_tree_layer_nxt, N, basis_len
    );
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        free(flattened_codeword);
        free(flattened_codeword_nxt);
        // free(flattened_tree_layer);
        //free(flattened_eval_basis);
        return;
    }
    
    compute_tree_layers<<<num_blocks * 2, threads_per_block>>> (
        device_codeword_nxt, device_layer_hashes, device_tree_layer, device_tree_layer_nxt, device_combined_sibling_codewords, device_concat_codeword_to_hash, device_digest, N
    );
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        free(flattened_codeword);
        free(flattened_codeword_nxt);
        // free(flattened_tree_layer);
        //free(flattened_eval_basis);
        return;
    }

    hipMemcpy(flattened_codeword_nxt, device_codeword_nxt, (N / 2) * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp1, device_temp1, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp2, device_temp2, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp3, device_temp3, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp4, device_temp4, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_temp5, device_temp5, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_alpha_offset, device_alpha_offset, N/2 * FIELD_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < N / 2; ++i) {
        for (int j = 0; j < FIELD_WORDS; ++j) {
            int index = i * FIELD_WORDS + j;
            if (index >= (N / 2) * FIELD_WORDS) {
                printf("Out-of-bounds access at index: %d\n", index);
                break;
            }
            codeword_nxt[i][j] = flattened_codeword_nxt[index];
        }
    }

    printf("First few codeword_nxt values:\n");
    for (int i = 0; i < 10; i++) {
        printf("%016lx ", flattened_codeword_nxt[i]);
    }
    printf("\n");
    hipFree(device_codeword);

    for (int i = 0; i < N / 2; ++i) {
        for (int j = 0; j < CONCAT_WORDS; ++j) {
            int index = i * CONCAT_WORDS + j;
            if (index >= (N / 2) * CONCAT_WORDS) {
                printf("Out-of-bounds access at index: %d\n", index);
                break;
            }
            tree_layer_nxt[i][j] = flattened_tree_layer_nxt[index];
        }
    }

    printf("First few tree_layer_nxt values:\n");
    for (int i = 0; i < 10; i++) {
        printf("%016lx ", flattened_tree_layer_nxt[i]);
    }
    printf("\n");
    hipFree(device_tree_layer);
    hipFree(device_tree_layer_nxt);
    free(flattened_tree_layer_nxt);
    // write_to_file("temp1.txt", flattened_temp1, FIELD_WORDS, N/2);
    // write_to_file("temp2.txt", flattened_temp2, FIELD_WORDS, N/2);
    // write_to_file("temp3.txt", flattened_temp3, FIELD_WORDS, N/2);
    // write_to_file("temp4.txt", flattened_temp4, FIELD_WORDS, N/2);
    // write_to_file("temp5.txt", flattened_temp5, FIELD_WORDS, N/2);
    // write_to_file("alpha_offset.txt", flattened_alpha_offset, FIELD_WORDS, N/2);

    // if (N == 64) { 
    //     int tree_idx = 2 * (int)log2(N);
    //     tree[tree_idx] = (uint64_t **)malloc((N/2) * sizeof(uint64_t *));
    //     for(int i = 0; i < N/2; i ++){
    //         tree[tree_idx][i] = (uint64_t *)malloc(CONCAT_WORDS * sizeof(uint64_t));
    //     }
    //     tree_idx++; 
    //     int next_N = N / 2;
    //     for(int layer = tree_idx; layer <= 18; layer++){
    //         tree[layer] = (uint64_t **)malloc((next_N / 2) * sizeof(uint64_t *));
    //         for(int i = 0; i < next_N / 2; i++){
    //             tree[layer][i] = (uint64_t *)malloc(2 * HASH_WORDS * sizeof(uint64_t));
    //             if (!tree[layer][i]) {
    //                 printf("Memory allocation failed at tree[%d][%d]\n", layer, i);
    //                 exit(1); 
    //             }
    //         }
    //         next_N = next_N / 2;
    //         printf("Allocated memory for tree[%d] with %d elements\n", layer, (next_N / 2) * HASH_WORDS);
    //     }
    
    //     int index = 13;
    //     int threads_per_block = 32;
    //     int num_blocks = (N / 2 + threads_per_block - 1) / threads_per_block;
    
    //     // Copy the last flattened layer from device to host
    //     hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N / 2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    
    //     // Unflatten and assign to tree[12]
    //     for (int i = 0; i < N / 2; i++) {
    //         for (int j = 0; j < CONCAT_WORDS; j++) {
    //             tree[12][i][j] = flattened_tree_layer_nxt[i * CONCAT_WORDS + j];
    //         }
    //     }
    
    //     free(flattened_tree_layer_nxt);
    //     hipFree(device_tree_layer_nxt);
    
    //     printf("Starting iterative Merkle kernel computation up the tree\n");
    //     N /= 2;
    
    //     // Allocate for iterative usage
    //     flattened_tree_layer_nxt = (uint64_t *)malloc((N / 2) * CONCAT_WORDS * sizeof(uint64_t));
    //     hipMalloc((void **)&device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t));
    
    //     while (N > 1) {
    //         int tpb = N / 2;
    //         int nb = (N + tpb - 1) / tpb;
    
    //         merkle_kernel<<<nb, tpb>>>(device_layer_hashes, device_merkle_root, device_tree_layer, device_tree_layer_nxt, N);
    //         hipDeviceSynchronize();
    
    //         // Copy next layer from device to host
    //         hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N / 2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    
    //         for (int i = 0; i < N / 2; i++) {
    //             for (int j = 0; j < HASH_WORDS; j++) {
    //                 tree[index][i][j] = flattened_tree_layer_nxt[i * HASH_WORDS + j];
    //             }
    //         }
    
    //         if (N == 2) {
    //             hipMemcpy(root, device_merkle_root, HASH_SIZE, hipMemcpyDeviceToHost);
    //         }
    
    //         N /= 2;
    //         index++;
    //     }
    
    //     free(flattened_tree_layer_nxt);
    //     hipFree(device_tree_layer_nxt);
    // }
    //something's up here
    if (N == 64) { 
        int tree_idx = 12;  //start with layer 12 for N == 64
        int next_N = N / 2; //initialize to 32 for the next layer size
        
        //allocate flattened memory for device -> host transfer
        uint64_t *flattened_tree_layer_nxt = (uint64_t *)malloc((N / 2) * CONCAT_WORDS * sizeof(uint64_t));
    
        //step 1: unflatten tree_layer_nxt computed by commit_kernel and assign it to tree[12]
        hipMalloc((void **)&device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t));
        hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N / 2) * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
    
        tree[tree_idx] = (uint64_t **)malloc((next_N) * sizeof(uint64_t *));
        for (int i = 0; i < next_N; i++) {
            tree[tree_idx][i] = (uint64_t *)malloc(CONCAT_WORDS * sizeof(uint64_t));
            for (int j = 0; j < CONCAT_WORDS; j++) {
                tree[tree_idx][i][j] = flattened_tree_layer_nxt[i * CONCAT_WORDS + j];
            }
        }
        tree_idx++;  // Move to the next tree layer index

        // Step 2: Assign tree_layer_nxt to tree_layer for the upcoming Merkle kernel computation
        hipMemcpy(device_tree_layer, device_tree_layer_nxt, (next_N / 2) * CONCAT_WORDS * sizeof(uint64_t), hipMemcpyDeviceToDevice);
        // N = N/2;//32
        // //step3: loop over the remaining layers and only compute hashes.
        // while(N>=2){ //reduce until N becomes 2
        //     int tpb = N;
        //     if(tpb == 0) {tpb = 1;}
        //     int nb = (N + (tpb - 1)) / tpb;

        //     //launch merkle_kernel which will prallely compute the merkle tree layers for each layer. start when layer has 32 elements
        //     merkle_kernel<<<nb, tpb>>>(device_layer_hashes, device_merkle_root, device_tree_layer, device_tree_layer_nxt, device_combined_sibling_codewords, device_digest, device_combined_sibling_hashes, N);
        //     hipDeviceSynchronize();

        //     //copy the next layer in the tree from device to host
        //     hipMalloc((void **)&device_tree_layer_nxt, (N / 2) * HASH_WORDS * sizeof(uint64_t)); //next layer in devicewould have N/2 elements
        //     hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (N/2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToDevice); //copy to device_layer_nxt
        //     printf("First few tree_layer_nxt values in the looooop:\n");
        //     for (int i = 0; i < HASH_WORDS; i++) {
        //         printf("%016lx ", flattened_tree_layer_nxt[i]);
        //     }
        //     printf("\n");
        //     //unflatten and store tree layer values in 'tree'
        //     tree[tree_idx] = (uint64_t **)malloc((next_N / 2) * sizeof(uint64_t *));
        //     for (int i = 0; i < next_N / 2; i++) {
        //         tree[tree_idx][i] = (uint64_t *)malloc(HASH_WORDS * sizeof(uint64_t));
        //         for (int j = 0; j < HASH_WORDS; j++) {
        //             tree[tree_idx][i][j] = flattened_tree_layer_nxt[i * HASH_WORDS + j];
        //         }
        //     }
        //     //update tree_idx? 
        // }
    
        //step 3: Loop over remaining layers, updating tree[layer] with each iteration
        while (next_N >= 2) {
            int tpb = min(32, next_N / 2);
            if(tpb == 0) {tpb = 1;}
            int nb = (next_N + tpb - 1) / tpb;
    
            // merkle_kernel for each layer (computes next layer hashes)
            merkle_kernel<<<nb, tpb>>>(device_layer_hashes, device_merkle_root, device_tree_layer, device_tree_layer_nxt, device_combined_sibling_codewords, device_digest, device_combined_sibling_hashes, next_N);
            hipDeviceSynchronize();
            
            hipMalloc((void **)&device_tree_layer_nxt, (next_N / 2) * HASH_WORDS * sizeof(uint64_t));
            hipMemcpy(flattened_tree_layer_nxt, device_tree_layer_nxt, (next_N / 2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
            printf("First few tree_layer_nxt values in the looooop:\n");
            for (int i = 0; i < HASH_WORDS + 4; i++) {
                printf("%016lx ", flattened_tree_layer_nxt[i]);
            }
            printf("\n");
            //unflatten and store in tree[tree_idx]
            tree[tree_idx] = (uint64_t **)malloc((next_N / 2) * sizeof(uint64_t *));
            for (int i = 0; i < next_N / 2; i++) {
                tree[tree_idx][i] = (uint64_t *)malloc(HASH_WORDS * sizeof(uint64_t));
                for (int j = 0; j < HASH_WORDS; j++) {
                    tree[tree_idx][i][j] = flattened_tree_layer_nxt[i * HASH_WORDS + j];
                }
            }
            printf("Populated tree[%d] with %d elements\n", tree_idx, next_N / 2);
            tree_idx++;
            
            //step 4: Update device_tree_layer with the contents of device_tree_layer_nxt
            hipMemcpy(device_tree_layer, device_tree_layer_nxt, (next_N / 2) * HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToDevice);

            next_N /= 2;
        // Handle the final layer (tree[16]) to compute the Merkle root
        if (next_N == 2) {
            // Copy Merkle root directly from device after merkle_kernel execution
            hipMemcpy(root, device_merkle_root, HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);
        
            // Print the computed Merkle root
            printf("Computed Merkle Root: ");
            for (int i = 0; i < HASH_WORDS; i++) {
                printf("%016lx ", root[i]);
            }
            printf("\n");
        }
        // if (next_N == 2) {
        //     // Allocate memory for the Merkle root on the device
        //     uint64_t *device_merkle_root;
        //     hipMalloc((void **)&device_merkle_root, HASH_WORDS * sizeof(uint64_t));

        //     // Launch kernel to compute the Merkle root
        //     compute_merkle_root_kernel<<<1, 1>>>(
        //         device_tree_layer,     // Input: Final layer with two sibling hashes
        //         device_merkle_root     // Output: Merkle root
        //     );
        //     hipDeviceSynchronize();

        //     // Copy the Merkle root from device to host
        //     hipMemcpy(root, device_merkle_root, HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);

        //     // Free device memory for the root
        //     hipFree(device_merkle_root);

        //     // Print the computed Merkle root
        //     printf("Computed Merkle Root: ");
        //     for (int i = 0; i < HASH_WORDS; i++) {
        //         printf("%016lx ", root[i]);
        //     }
        //     printf("\n");
        // }
        } //close while loop
        free(flattened_tree_layer_nxt);
    }

    
    
    hipFree(device_codeword_nxt);
    hipFree(device_alpha);
    hipFree(device_offset);
    hipFree(device_denominator_inv);
    hipFree(device_eval_basis);
    hipFree(device_merkle_root);
    hipFree(device_layer_hashes);

    free(flattened_codeword);
    free(flattened_codeword_nxt);

    printf("Memory freed and commit_launch completed.\n");
}
