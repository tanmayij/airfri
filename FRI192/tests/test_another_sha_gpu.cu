#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include "../include/hash.cuh"  // Include the GPU hash function

#define HASH_WORDS 4  // SHA3-256 output size in uint64_t words

// Function prototype for the SHA3 GPU implementation
__device__ void SHA3(uint8_t *hm, const uint8_t *msg, size_t msg_len, size_t bitSize);

__global__ void sha3_kernel(uint64_t *input_value, uint64_t *output_hash, size_t input_size) {
    uint8_t msg[16];  // Assuming 2 uint64_t inputs (16 bytes)
    memcpy(msg, input_value, input_size);

    // Compute SHA3-256 hash
    SHA3((uint8_t *)output_hash, msg, input_size, 256);
}

// Function to check SHA3 hash on GPU
int check_sha3_hash_gpu(uint64_t *input_value, uint64_t *expected_hash, size_t input_size) {
    uint64_t *d_hash, h_hash[HASH_WORDS];  // SHA3-256 output (4 uint64_t values)
    uint64_t *d_input;

    // Allocate device memory
    hipMalloc((void **)&d_input, input_size);
    hipMalloc((void **)&d_hash, HASH_WORDS * sizeof(uint64_t));

    // Copy input to device
    hipMemcpy(d_input, input_value, input_size, hipMemcpyHostToDevice);

    // Launch SHA3 kernel (single thread, as SHA3 is not parallelized)
    sha3_kernel<<<1, 1>>>(d_input, d_hash, input_size);
    hipDeviceSynchronize();

    // Copy hash result back to host
    hipMemcpy(h_hash, d_hash, HASH_WORDS * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Print input values for debugging
    printf("GPU Input Values: ");
    for (size_t i = 0; i < input_size / sizeof(uint64_t); i++) {
        printf("%016lx ", input_value[i]);
    }
    printf("\n");

    // Print computed SHA3 hash as uint64_t
    printf("GPU Computed SHA3-256 Hash (as uint64_t): ");
    for (size_t i = 0; i < HASH_WORDS; i++) {
        printf("%016lx ", h_hash[i]);
    }
    printf("\n");

    // Compare computed hash with expected hash
    int result = memcmp(expected_hash, h_hash, HASH_WORDS * sizeof(uint64_t)) == 0;

    // Free device memory
    hipFree(d_input);
    hipFree(d_hash);

    return result;
}

// Example usage
int main() {
    // Define sizes
    size_t input_size = 16 * sizeof(uint64_t);  // 16 uint64_t values (128 bytes)
    size_t hash_size = 4 * sizeof(uint64_t);    // SHA3-256 output (32 bytes)

    // Allocate memory for input values
    uint64_t *input_value = (uint64_t *)malloc(input_size);
    if (input_value == NULL) {
        fprintf(stderr, "Memory allocation failed for input_value\n");
        exit(1);
    }

    // Assign values to input dynamically

    input_value[0] = 0xea09e462003c9a76;
    input_value[1] = 0x7f6202591b6d54ab;
    input_value[2] = 0xb5f1ebb26fe44a2f;
    input_value[3] = 0x5701e630a806778c;
    input_value[4] = 0x1ecd7a4ab500524e;
    input_value[5] = 0x890005156c5daab9;
    input_value[6] = 0x484ee268faa0f3af;
    input_value[7] = 0x732f3b116510ec16;
    input_value[8] = 0xb5482ae7600392bf;
    input_value[9] = 0x255dbd5446ab15c8;
    input_value[10] = 0x3f1476e9a98a36f1;
    input_value[11] = 0x364600387d9e9355;
    input_value[12] = 0x40f4af00a3929a87;
    input_value[13] = 0xe3044a26718c63a2;
    input_value[14] = 0x7d3e2f4460b1628e;
    input_value[15] = 0x8d74c72527cc8b27;

    // Allocate memory for expected hash
    uint64_t *expected_hash = (uint64_t *)malloc(hash_size);
    if (expected_hash == NULL) {
        fprintf(stderr, "Memory allocation failed for expected_hash\n");
        free(input_value);
        exit(1);
    }

    // Assign expected hash values
    expected_hash[0] = 0x807ef22d564090aa;
    expected_hash[1] = 0x192c3385ff3473df;
    expected_hash[2] = 0xc54461a269b5e70b;
    expected_hash[3] = 0x64cb44f4f0e32af0;

    // Check SHA3 hash computation on GPU
    int is_match = check_sha3_hash_gpu(input_value, expected_hash, input_size);

    // Print the result
    if (is_match) {
        printf("GPU Hash matches the expected output!\n");
    } else {
        printf("GPU Hash does NOT match the expected output!\n");
    }

    // Free allocated memory
    free(input_value);
    free(expected_hash);

    return 0;
}